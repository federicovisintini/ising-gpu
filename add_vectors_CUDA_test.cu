
#include <hip/hip_runtime.h>
#include<iostream>

#define N 65535

__global__ void add(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate memory on GPU
    hipMalloc( &dev_a, N * sizeof(int) );
    hipMalloc( &dev_b, N * sizeof(int) );
    hipMalloc( &dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // display the result
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
